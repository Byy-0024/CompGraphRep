#include "hip/hip_runtime.h"
#include "pr.cuh"

using namespace std;

#define DEBUG(x)
#define N_THREADS_PER_BLOCK (1 << 7)

__global__
void compute_an_iteration(cuda_csr_graph *d_g, float *pr_val, float *prev_pr_val) {
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < d_g->num_nodes) {
		pr_val[tid] = 0;
        for (int i = d_g->d_edgesOffset[tid]; i < d_g->d_edgesOffset[tid] + d_g->d_edgesSize[tid]; ++i) {
            node v = d_g->d_adjacencyList[i];
            pr_val[tid] += prev_pr_val[v]; 
        }
    }
}

__global__
void get_deg(cuda_hvi_graph *d_g, int *deg) {
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < d_g->num_nodes) {
		int res = 0;
		for (int i = d_g->d_HVIOffset[tid]; i < d_g->d_HVIOffset[tid+1];) {
            HybridVertexInterval v = d_g->d_HVIList[i];
            if (v & HVI_LEFT_BOUNDARY_MASK) {
                node left = v & GETNODE_MASK;
                node right = d_g->d_HVIList[i+1] & GETNODE_MASK;
                res += (right - left + 1);
                i += 2;
            }
            else {
                res++;
                i++;
            }
        }
		deg[tid] = res;
	}
}

__global__
void scale_by_deg(int n, float *pr_val, int *deg) {
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n) {
		pr_val[tid] /= (float) deg[tid];
	}
}

__global__
void compute_an_iteration(cuda_hvi_graph *d_g, float *pr_val, float *prev_pr_val) {
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < d_g->num_nodes) {
		pr_val[tid] = 0;
        for (int i = d_g->d_HVIOffset[tid]; i < d_g->d_HVIOffset[tid+1];) {
            HybridVertexInterval v = d_g->d_HVIList[i];
            if (v & HVI_LEFT_BOUNDARY_MASK) {
                node left = v & GETNODE_MASK;
                node right = d_g->d_HVIList[i+1] & GETNODE_MASK;
                for (node next = left; next <= right; next++) {
                    pr_val[tid] += prev_pr_val[next];
                }
                i += 2;
            }
            else {
                pr_val[tid] += prev_pr_val[v];
                i++;
            }
        }
	}
}

__global__
void compute_difference(const int n, float *pr_val, float *prev_pr_val, int *d_deg, float *d_diff) {
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float sdata[N_THREADS_PER_BLOCK];
	float t_res = 0;
	if (tid < n) {
		if (d_deg[tid] > 0)	t_res = fabs(pr_val[tid] - prev_pr_val[tid] * d_deg[tid]);
	}

	const int local_idx = threadIdx.x;
	sdata[local_idx] = t_res;
	__syncthreads();

	for (unsigned int s = N_THREADS_PER_BLOCK / 2; s > 0; s >>= 1) {
		if (local_idx < s) {
			sdata[local_idx] += sdata[local_idx + s];
		}
		__syncthreads();
	}

	if (local_idx == 0) {
		atomicAdd(d_diff, sdata[0]);
	}
}

void pageRank(cuda_csr_graph &G, vector<float> &pr_val, int max_iter, float epsilon) {
	const int n = G.num_nodes;
	const int n_blocks = (n + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK;

    cuda_csr_graph *d_g;
	float *d_pr_val; 
    float *d_prev_pr_val;
	float *d_diff;
	int iter = 0;
	float diff = (float) 1;

    hipMalloc((void**)&d_g, sizeof(cuda_csr_graph));
	hipMalloc((void **)&d_pr_val, n * sizeof(float));
	hipMalloc((void **)&d_prev_pr_val, n * sizeof(float));
	hipMalloc((void **)&d_diff, sizeof(float));
	
    hipMemcpy(d_g, &G, sizeof(cuda_csr_graph), hipMemcpyHostToDevice);
	hipMemcpy(d_pr_val, pr_val.data(), n * sizeof(float), hipMemcpyHostToDevice);

	while (iter < max_iter) {
	// while (iter < max_iter && diff > epsilon) {
        hipMemcpy(d_prev_pr_val, d_pr_val, n * sizeof(float), hipMemcpyDeviceToDevice);
		scale_by_deg <<<n_blocks, N_THREADS_PER_BLOCK>>> (n, d_prev_pr_val, G.d_edgesSize);
		errorCheck(hipDeviceSynchronize(), "scaling by degree");
		compute_an_iteration <<<n_blocks, N_THREADS_PER_BLOCK>>> (d_g, d_pr_val, d_prev_pr_val);
		errorCheck(hipDeviceSynchronize(), "updating pr_val");
		hipMemset(&d_diff, 0, sizeof(float));
		errorCheck(hipDeviceSynchronize(), "initializing difference");
		compute_difference<<<n_blocks, N_THREADS_PER_BLOCK>>> (n, d_pr_val, d_prev_pr_val, G.d_edgesSize, d_diff);
		errorCheck(hipDeviceSynchronize(), "computing difference");
		hipMemcpy(&diff, d_diff, sizeof(float), hipMemcpyDeviceToHost);
		// printf("iter %d, diff %.3f\n", iter, diff);
		++iter;
	}

	hipMemcpy(pr_val.data(), d_pr_val, n * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_g);
	hipFree(d_pr_val);
    hipFree(d_prev_pr_val);
	hipFree(&d_diff);
}

void pageRank(cuda_hvi_graph &G, vector<float> &pr_val, int max_iter, float epsilon) {
	const int n = G.num_nodes;
	const int n_blocks = (n + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK;

	cuda_hvi_graph *d_g;
    float *d_pr_val; 
    float *d_prev_pr_val;
	int *d_deg;
	float *d_diff;
	int iter = 0;
	float diff = 1;

	hipMalloc((void**)&d_g, sizeof(cuda_hvi_graph));
    hipMalloc((void **)&d_pr_val, n * sizeof(float));
	hipMalloc((void **)&d_prev_pr_val, n * sizeof(float));
	hipMalloc((void **)&d_deg, n * sizeof(int));
	hipMalloc((void **)&d_diff, sizeof(float));
    hipMemcpy(d_g, &G, sizeof(cuda_hvi_graph), hipMemcpyHostToDevice);
	hipMemcpy(d_pr_val, pr_val.data(), n * sizeof(float), hipMemcpyHostToDevice);

	get_deg <<<n_blocks, N_THREADS_PER_BLOCK>>> (d_g, d_deg);

	while (iter < max_iter && diff > epsilon) {
        hipMemcpy(d_prev_pr_val, d_pr_val, n * sizeof(float), hipMemcpyDeviceToDevice);
		scale_by_deg <<<n_blocks, N_THREADS_PER_BLOCK>>> (n, d_prev_pr_val, d_deg);
		errorCheck(hipDeviceSynchronize(), "scaling by degree");
		compute_an_iteration <<<n_blocks, N_THREADS_PER_BLOCK>>> (d_g, d_pr_val, d_prev_pr_val);
		errorCheck(hipDeviceSynchronize(), "updating pr_val");
		hipMemset(&d_diff, 0, sizeof(float));
		compute_difference<<<n_blocks, N_THREADS_PER_BLOCK>>> (n, d_pr_val, d_prev_pr_val, d_deg, d_diff);
		errorCheck(hipDeviceSynchronize(), "computing difference");
		hipMemcpy(&diff, d_diff, sizeof(float), hipMemcpyDeviceToHost);
		++iter;
	}

	// Copying output back to host
	hipMemcpy(pr_val.data(), d_pr_val, n * sizeof(float), hipMemcpyDeviceToHost);

	// Cleanup
	hipFree(d_g);
	hipFree(&d_diff);
	hipFree(d_deg);
	hipFree(d_pr_val);
    hipFree(d_prev_pr_val);
}