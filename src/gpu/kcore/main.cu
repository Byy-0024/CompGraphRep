#include <iostream>
#include <chrono>

#include "../../Graph.hpp"
#include "../../Interval.hpp"
#include "kcore.cuh"

using namespace std;

void print(vector<int> &v, int print_size) {
	cout << "{ ";
	for (int i = 0; i < print_size; ++i) {
		cout << v[i];
		if (i < print_size - 1)
			cout << ", ";
	}
	cout << "... }" << endl;
}

class Checker {
	vector<int> expected_answer;
public:
	Checker(vector<int> exp_ans): expected_answer(exp_ans) {}

	pair<int, int> count_visited_vertices(const vector<int> &core_numbers) {
		int depth = 0;
		int count = 0;
		for (int x : core_numbers) {
			if (x < INT_MAX) {
				++count;
				if (x > depth) {
					depth = x;
				}
			}
		}
		return {count, depth};
	}

	bool check(vector<int> answer) {
		assert(answer.size() == expected_answer.size());
		bool is_ok = true;
		int position_wrong = -1;
		for (int i = 0; i < answer.size(); ++i) {
			if (answer.at(i) != expected_answer.at(i)) {
				is_ok = false;
				position_wrong = i;
				break;
			}
		}
        // return is_ok;
		if (is_ok) {
			// pair<int, int> graph_output = count_visited_vertices(answer);
			// int n_visited_vertices = graph_output.first;
			// int depth = graph_output.second;
			// printf("CHECKED SUCCESSFULY! Number of visited vertices: %i, depth: %i \n", n_visited_vertices, depth);
            return true;
		}
		else {
			printf("Something went wrong!\n");
			printf("Answer at %i equals %i but should be equal to %i\n", position_wrong, answer[position_wrong], expected_answer[position_wrong]);
            return false;
		}
	}
};

void read_ordering(string orderingdict, vector<node> &origin_to_new, int num_nodes) {
    ifstream input(orderingdict);
    string line;
    origin_to_new.clear();
    origin_to_new.resize(num_nodes, UINT_MAX);
    while (getline(input, line)) {
        int u, v;
        stringstream(line).ignore(0, ' ') >> u >> v;
        origin_to_new[u] = v;
    }
}

void cuda_kcore_test(string graphdict) {
    time_counter _t;
    _t.start();
    Graph G(graphdict + "/origin/csr_vlist.bin", graphdict + "/origin/csr_elist.bin");
    _t.stop();
    _t.print("graph construnction");
    HVI G_hvi(G);
    size_t _size = G_hvi.size_in_bytes();
    printf("Size of new HVI Graph: %.3f MB!\n", (float)_size / (1 << 20));
    
    vector<int> core_numbers;
    time_counter _t_csr, _t_hvi;

    G.get_degs(core_numbers);
    _t_csr.start();
    cuda_csr_graph G_cuda;
    G_cuda.init(G);
    // _t_csr.start();
    kcoreGPU(G_cuda, core_numbers);
    // _t_csr.stop();
    G_cuda.free();
    _t_csr.stop();
    Checker checker(core_numbers);
    _t_csr.print("GPU k-core decomposition using CSR");
    print(core_numbers, 10);

    G.get_degs(core_numbers);
    _t_hvi.start();
    cuda_hvi_graph G_cuda_run;
    G_cuda_run.init(G_hvi);
    // _t_hvi.start();
    kcoreGPU(G_cuda_run, core_numbers);
    // _t_hvi.stop();
    G_cuda_run.free();
    _t_hvi.stop();
    _t_hvi.print("GPU k-core decomposition using HVI");
    print(core_numbers, 10);
    
    if (checker.check(core_numbers)) printf("K-core decomposition verification: Pass!\n");
    else printf("K-core decomposition verification: Fail!\n");
}

void cuda_kcore_test(string graphdict, string orderingmethod) {
    time_counter _t;
    vector<int> core_numbers;
    time_counter _t_csr, _t_comp, _t_hvi;
    
    // K-core decomposition using CSR graph format.
    _t.start();
    Graph G(graphdict + "/" + orderingmethod + "/csr_vlist.bin", graphdict + "/" + orderingmethod + "/csr_elist.bin");
    _t.stop();
    _t.print("csr graph construnction");
    size_t _size = G.size_in_bytes();
    printf("Size of new CSR Graph: %.3f MB!\n", (float)_size / (1 << 20));
    G.get_degs(core_numbers);
    _t_csr.start();
    cuda_csr_graph G_cuda;
    G_cuda.init(G);
    kcoreGPU(G_cuda, core_numbers);
    G_cuda.free();
    _t_csr.stop();
    Checker checker(core_numbers);
    _t_csr.print("GPU k-core decomposition using CSR");
    print(core_numbers, 10);

    // K-core decomposition using CompressGraph format.
    _t.start();
    Graph G_comp(graphdict + "/compress/csr_vlist.bin", graphdict + "/compress/csr_elist.bin");
    _t.stop();
    _t.print("CompressGraph construnction");
    _size = G_comp.size_in_bytes();
    printf("Size of CompressGraph Graph: %.3f MB!\n", (float)_size / (1 << 20));
    G_comp.get_degs(core_numbers);
    _t_comp.start();
    cuda_csr_graph G_cuda_comp;
    G_cuda_comp.init(G_comp);
    kcoreGPU(G_cuda_comp, core_numbers);
    G_cuda_comp.free();
    _t_comp.stop();
    _t_comp.print("GPU k-core decomposition using CSR");
    // print(core_numbers, 10);

    // K-core decomposition using HVI graph format. 
    _t.clean();
    _t.start();
    HVI G_hvi(graphdict + "/" + orderingmethod + "/hvi_offsets.bin", graphdict + "/" + orderingmethod + "/hvi_list.bin");
    _t.stop();
    _t.print("hvi graph construnction");
    _size = G_hvi.size_in_bytes();
    printf("Size of new HVI Graph: %.3f MB!\n", (float)_size / (1 << 20));

    G.get_degs(core_numbers);
    _t_hvi.start();
    cuda_hvi_graph G_cuda_run;
    G_cuda_run.init(G_hvi);
    kcoreGPU(G_cuda_run, core_numbers);
    G_cuda_run.free();
    _t_hvi.stop();
    _t_hvi.print("GPU k-core decomposition using HVI");
    print(core_numbers, 10);

    if (checker.check(core_numbers)) printf("K-core decomposition verification: Pass!\n");
    else printf("K-core decomposition verification: Fail!\n");
}

// Tests speed of a BFS algorithm
int main(int argc, char **argv) {  
	string graphdict = argv[1];
    string orderingmethod = argv[2];
    int device_id = stoi(argv[3]);
    hipSetDevice(device_id);
    printf("======= %s Ordering ===========\n", orderingmethod.c_str());
    cuda_kcore_test(graphdict, orderingmethod);
    // printf("======= Greedy Ordering ===========\n");
    // cuda_bfs_test(graphdict, orderingdict, nodequerydict);
	return 0;
}
